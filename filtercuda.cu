#include "hip/hip_runtime.h"
/*Compile: nvcc -o cuda filtercuba.cu*/

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <fcntl.h>
#include <unistd.h>
#include <hip/hip_runtime.h>

#define FRACTION_CEILING(numerator, denominator) ((numerator+denominator-1)/denominator)
#define BLOCK_SIZE  24

unsigned char* openGreyImage(char *path, unsigned int height, unsigned int width);
unsigned char** openRGBImage(char *path, unsigned int height, unsigned int width);
void saveGreyImage(char *path, unsigned char *image_array, unsigned int height, unsigned int width);
void saveRGBImage(char *path, unsigned char **image_array, unsigned int height, unsigned int width);
__global__ void convolution(unsigned char *in_image, unsigned char *out_image, int height, int width, int *cfilter);

/*
 * Open the images, process it and save it
 */
int main(int argc, char **argv) {
	int i, j, n, is_RGB, filter[9] = {1, 2, 1, 2, 4, 2, 1, 2, 1}, *cfilter;
	unsigned int width, height;
	float elapsed_time;
	char *input_path, *output_path;
	unsigned char *Grey_image_array, **RGB_image_array, *tempGrey, **tempRGB, *in_image, *out_image;
	hipEvent_t start, stop;	

	//if arguments is OK
	if (argc < 5 || argc > 6) {
		fprintf(stderr, "Not correct arguments. Please give me 1) the image file path, 2) the width and 3) the height of image, 4) 0 or 1 if it's Grey or RGB and 5) optinal the number of times that the filter will be applied.\n");
		exit(EXIT_FAILURE);
	}

	input_path = argv[1];
	width = atoi(argv[2]);
	height = atoi(argv[3]);
	is_RGB = atoi(argv[4]);
	if(argc == 6)
		n = atoi(argv[5]);
	else
		n = -1;

	//open the image
	if(!is_RGB)
		Grey_image_array = openGreyImage(input_path, height, width);
	else
		RGB_image_array = openRGBImage(input_path, height, width);
	
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE,1);
	dim3 dimGrid(FRACTION_CEILING(width, BLOCK_SIZE),FRACTION_CEILING(height, BLOCK_SIZE),1);

	hipMalloc((void**)&in_image, height * width * sizeof(unsigned char));
	hipMalloc((void**)&out_image, height * width * sizeof(unsigned char));
	hipMalloc((void**)&cfilter, 9 * sizeof(int));

	hipMemcpy(cfilter, filter, 9 * sizeof(int), hipMemcpyHostToDevice);
	
	//time starts
	hipEventCreate(&start);
	hipEventRecord(start,0);
	
	if(n < 0){
		//repeat till the two array is equal, use memcpy so that cpu compare the arrays
		i = 0;
		if(!is_RGB){
			tempGrey = (unsigned char*)malloc(height * width * sizeof(unsigned char));
			do{
				if(i % 2 == 0){
					hipMemcpy(in_image, Grey_image_array, height * width * sizeof(unsigned char), hipMemcpyHostToDevice);
					convolution<<<dimGrid,dimBlock>>>(in_image, out_image, height, width, cfilter);
					hipDeviceSynchronize();
					hipMemcpy(tempGrey, out_image, height * width * sizeof(unsigned char), hipMemcpyDeviceToHost);
				}else{
					hipMemcpy(in_image, tempGrey, height * width * sizeof(unsigned char), hipMemcpyHostToDevice);
					convolution<<<dimGrid,dimBlock>>>(in_image, out_image, height, width, cfilter);
					hipDeviceSynchronize();
					hipMemcpy(Grey_image_array, out_image, height * width * sizeof(unsigned char), hipMemcpyDeviceToHost);
				}
				i++;
			}while(memcmp(Grey_image_array, tempGrey, height * width) != 0);
			free(tempGrey);
		}else{
			tempRGB = (unsigned char**)malloc(3 * sizeof(unsigned char*));
			for(j = 0; j < 3; j++)
				tempRGB[j] = (unsigned char*)malloc(height * width * sizeof(unsigned char));
			do{
				if(i % 2 == 0){
					for(j = 0; j < 3; j++){
						hipMemcpy(in_image, RGB_image_array[j], height * width * sizeof(unsigned char), hipMemcpyHostToDevice);
						convolution<<<dimGrid,dimBlock>>>(in_image, out_image, height, width, cfilter);
						hipDeviceSynchronize();
						hipMemcpy(tempRGB[j], out_image, height * width * sizeof(unsigned char), hipMemcpyDeviceToHost);
					}
				}else{
					for(j = 0; j < 3; j++){
						hipMemcpy(in_image, tempRGB[j], height * width * sizeof(unsigned char), hipMemcpyHostToDevice);
						convolution<<<dimGrid,dimBlock>>>(in_image, out_image, height, width, cfilter);
						hipDeviceSynchronize();
						hipMemcpy(RGB_image_array[j], out_image, height * width * sizeof(unsigned char), hipMemcpyDeviceToHost);
					}
				}
				i++;
			}while(memcmp(RGB_image_array[0], tempRGB[0], height * width) != 0 || memcmp(RGB_image_array[1], tempRGB[1], height * width) != 0 || memcmp(RGB_image_array[2], tempRGB[2], height * width) != 0);
			for(j = 0; j < 3; j++)
				free(tempRGB[j]);
			free(tempRGB);
		}
	}else{
		//repeat n times, use swap because no need to swap
		if(!is_RGB){
			hipMalloc((void**)&tempGrey, height * width * sizeof(unsigned char));
			hipMemcpy(in_image, Grey_image_array, height * width * sizeof(unsigned char), hipMemcpyHostToDevice);
			for(i = 0; i < n; i++){
				convolution<<<dimGrid,dimBlock>>>(in_image, out_image, height, width, cfilter);
				hipDeviceSynchronize();
				tempGrey = in_image;
				in_image = out_image;
				out_image = tempGrey;
			}
			hipMemcpy(Grey_image_array, out_image, height * width * sizeof(unsigned char), hipMemcpyDeviceToHost);
			hipFree(tempGrey);
		}else{
			tempRGB = (unsigned char**)malloc(3 * sizeof(unsigned char*));
			for(i = 0; i < 3; i++)
				hipMalloc((void**)&tempRGB[i], height * width * sizeof(unsigned char));
			for(i = 0; i < 3; i++){
				hipMemcpy(in_image, RGB_image_array[i], height * width * sizeof(unsigned char), hipMemcpyHostToDevice);
				for(j = 0; j < n; j++){
					convolution<<<dimGrid,dimBlock>>>(in_image, out_image, height, width, cfilter);
					hipDeviceSynchronize();
					tempRGB[i] = in_image;
					in_image = out_image;
					out_image = tempRGB[i];
				}
				hipMemcpy(RGB_image_array[i], out_image, height * width * sizeof(unsigned char), hipMemcpyDeviceToHost);
			}
			for(i = 0; i < 3; i++)
				hipFree(tempRGB[i]);
			free(tempRGB);
		}
	}

	//time finishes
	hipEventCreate(&stop);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	
	hipEventElapsedTime(&elapsed_time, start, stop);
	
	hipEventDestroy(start);
	hipEventDestroy(stop);
	
	printf("Finished for file %s after %f sec\n", input_path, elapsed_time/1000);
	
	//append "(convolution)" at the end of the filename
	output_path =  (char*) malloc((strlen(input_path) + 1 + strlen("(convolution)")) * sizeof(char));
	strncpy(output_path, input_path, strlen(input_path) - strlen(".raw"));
	strcat(output_path, "(convolution)");
	strcat(output_path, ".raw");
	
	//save image in another file
	if(!is_RGB)
		saveGreyImage(output_path, Grey_image_array, height, width);
	else
		saveRGBImage(output_path, RGB_image_array, height, width);

	//time to leave, destroy dynamic space
	if(!is_RGB)
		free(Grey_image_array);
	else{
		for(i = 0; i < 3; i++)
			free(RGB_image_array[i]);
		free(RGB_image_array);
	}
	hipFree(in_image);
	hipFree(out_image);
	hipFree(cfilter);

	//bye
	exit(EXIT_SUCCESS);
}


/*
 * Reads the Grey image file and store it into an array
 */
unsigned char* openGreyImage(char *path, unsigned int height, unsigned int width) {

	unsigned char *image_array = (unsigned char*) malloc(height * width * sizeof(unsigned char));

	FILE* image_file = fopen(path, "r");
	if (image_file == NULL) {
		perror("fopen failed");
		exit(EXIT_FAILURE);
	}

	if (fread(image_array, sizeof(unsigned char), height * width, image_file) != height * width) {
		fprintf(stderr, "fread failed");
		exit(EXIT_FAILURE);
	}

	fclose(image_file);

	return image_array;
}

/*
 * Reads the RGB image file and store each colour to an array
 */
unsigned char** openRGBImage(char *path, unsigned int height, unsigned int width){

	unsigned char *temp = (unsigned char*) malloc(3 * height * width * sizeof(unsigned char));

	FILE* image_file = fopen(path, "r");
	if (image_file == NULL) {
		perror("fopen failed");
		exit(EXIT_FAILURE);
	}

	if (fread(temp, sizeof(unsigned char), 3 * height * width, image_file) !=3* height * width) {
		fprintf(stderr, "fread failed");
		exit(EXIT_FAILURE);
	}

	fclose(image_file);

	unsigned int i;
	unsigned char **image_array = (unsigned char**) malloc(3 * sizeof(unsigned char*));
	for (i = 0; i < 3; i++){
		image_array[i] = (unsigned char*) malloc(height * width * sizeof(unsigned char));
	}
	for (i = 0; i < height * width * 3; i++){
		image_array[i % 3][i / 3] = temp[i];
	}
	free(temp);
	return image_array;
}

/*
 * Save Grey image to another image file specified by path
 */
void saveGreyImage(char *path, unsigned char *image_array, unsigned int height, unsigned int width) {
	
	FILE* image_file = fopen(path, "w");
	if (image_file == NULL) {
		perror("fopen failed");
		exit(EXIT_FAILURE);
	}

	if (fwrite(image_array, sizeof(unsigned char), height * width, image_file) != height * width) {
		fprintf(stderr, "fwrite failed");
		exit(EXIT_FAILURE);
	}

	fclose(image_file);

	return;

}

/*
 * Save RGB image to another image file specified by path
 */
void saveRGBImage(char *path, unsigned char **image_array, unsigned int height, unsigned int width){

	unsigned int i;
	unsigned char *temp = (unsigned char*) malloc(3 * height * width * sizeof(unsigned char));

	for (i = 0; i < height * width * 3; i++){
		temp[i] = image_array[i % 3][i / 3];
	}

	FILE* image_file = fopen(path, "w");
	if (image_file == NULL) {
		perror("fopen failed");
		exit(EXIT_FAILURE);
	}

	if (fwrite(temp, sizeof(unsigned char), height * width*3, image_file) != height * width*3) {
		fprintf(stderr, "fwrite failed");
		exit(EXIT_FAILURE);
	}

	fclose(image_file);

	free(temp);

	return;
}

/*
 * Function for gpu which apply the filter
 */
__global__ void convolution(unsigned char *in_image, unsigned char *out_image, int height, int width, int *cfilter ) {
	
	const unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int y = blockDim.y * blockIdx.y + threadIdx.y;

	if (y >= height || x >= width){
    	   return;
	}
    	

	int i, j, s = 1, y_idx, x_idx, sum = 0;

	for (i = -s; i <= s; i++) {
		for ( j = -s; j <= s; j++) {
			y_idx = y + i;
			x_idx = x + j;
			if (y_idx >= height || y_idx < 0 || x_idx >= width || x_idx < 0) {
				y_idx = y;
				x_idx = x;
			}
			sum += in_image[width*(y_idx)+(x_idx)] * cfilter[3*(i+1)+(j+1)];
		}	
	}

	out_image[width*y+x] =(unsigned char)((float)sum/16);	
}

